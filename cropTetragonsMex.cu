#include "hip/hip_runtime.h"

#include "mex.h"
#include "gpu/mxGPUArray.h"

#include <npp.h>

#include <math.h> 

#define MATLAB_ASSERT(expr,msg) if (!(expr)) { mexErrMsgTxt(msg);}

#if !defined(MX_API_VER) || MX_API_VER < 0x07030000
typedef size_t mwSize;
typedef size_t mwIndex;
#endif

void mexFunction(int nlhs, mxArray *plhs[], 
    int nrhs, const mxArray *prhs[])
{
	MATLAB_ASSERT( nrhs == 3, "cropTetragonsMex: Wrong number of input parameters: expected 3");
    MATLAB_ASSERT( nlhs == 1, "cropTetragonsMex: Wrong number of output arguments: expected 1");
	
	// Fix input parameter order:
	const mxArray *imInPtr = (nrhs >= 0) ? prhs[0] : NULL; // image
	const mxArray *tetragonInPtr = (nrhs >= 1) ? prhs[1] : NULL; // tetragons
	const mxArray *szInPtr = (nrhs >= 2) ? prhs[2] : NULL; // output image size
	
	// Fix output parameter order:
	mxArray **cropsOutPtr = (nlhs >= 1) ? &plhs[0] : NULL; // croped and resized patches
	
	// Get the image
	MATLAB_ASSERT(mxGetNumberOfDimensions(imInPtr) == 3, "cropTetragonsMex: the image is not 3-dimensional");
	MATLAB_ASSERT(mxGetClassID(imInPtr) == mxSINGLE_CLASS, "cropTetragonsMex: the image should be of type SINGLE");
	MATLAB_ASSERT(mxGetPi(imInPtr) == NULL, "cropTetragonsMex: image should not be complex");

    const mwSize* dimensions = mxGetDimensions(imInPtr);
	mwSize imageHeight = dimensions[0];
	mwSize imageWidth = dimensions[1];
	mwSize numChannels = dimensions[2];
	MATLAB_ASSERT(numChannels == 3, "cropTetragonsMex: image should contain 3 channels");

	float* imageData = (float*) mxGetData(imInPtr);

	// get tetragons
	MATLAB_ASSERT(mxGetNumberOfDimensions(tetragonInPtr) == 2, "cropTetragonsMex: <tetragons> input is not 2-dimensional");
	MATLAB_ASSERT(mxGetClassID(tetragonInPtr) == mxDOUBLE_CLASS, "cropTetragonsMex: <tetragons> input is not of type double");
	MATLAB_ASSERT(mxGetPi(tetragonInPtr) == NULL, "cropTetragonsMex: <tetragons> input should not be complex");
	MATLAB_ASSERT(mxGetN(tetragonInPtr) == 8, "cropTetragonsMex: <tetragons> input should be of size #tetragons x 8");
	
	mwSize numTetragon = mxGetM(tetragonInPtr);
	double* tetragonData = (double*) mxGetData(tetragonInPtr); // y1, x1, y2, x2, y3, x3, y4, x4

	// get output size
	MATLAB_ASSERT(mxGetNumberOfElements(szInPtr) == 2, "cropTetragonsMex: <outputSize> input should contain 2 numbers");
	MATLAB_ASSERT(mxGetClassID(szInPtr) == mxDOUBLE_CLASS, "cropTetragonsMex: <outputSize> input is not of type double");
	MATLAB_ASSERT(mxGetPi(szInPtr) == NULL, "cropTetragonsMex: <outputSize> input should not be complex");
	
	double* outputSizeData = (double*) mxGetData(szInPtr);
	int targetHeight = (int) (outputSizeData[0] + 0.5);
	int targetWidth = (int) (outputSizeData[1] + 0.5);

	// initialize GPU
	mxInitGPU();

	// copy image to the GPU
	mxGPUArray const *inputImage;
    float const *d_inputImage;
	inputImage = mxGPUCreateFromMxArray(imInPtr);
	d_inputImage = (float const *)(mxGPUGetDataReadOnly(inputImage));

	// allocate memory for the output
    mxGPUArray *outputData;
    float *d_outputData;
	const mwSize outputDimensions[4] = { targetHeight, targetWidth, numChannels, numTetragon };
	outputData = mxGPUCreateGPUArray(4, outputDimensions, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES) ; //MX_GPU_DO_NOT_INITIALIZE);
	d_outputData = (float *)(mxGPUGetData(outputData));

	// initialize some cropping arguments
	NppiSize nppiImageSize = {};
	nppiImageSize.width = imageHeight; // CAUTION: NPPI thinks that the image is transposed 
	nppiImageSize.height = imageWidth;

	int channelValueSize = sizeof(float);
	int imageStep = imageHeight * channelValueSize;
	int targetStep = targetHeight * channelValueSize;

    NppiRect targetRect = {};
    targetRect.x = 0;
    targetRect.y = 0;
    targetRect.width = targetHeight;
    targetRect.height = targetWidth;

    double aDstQuad[4][2] = { {0.0, 0.0}, {targetHeight - 1.0, 0.0}, {targetHeight - 1.0, targetWidth - 1.0}, {0.0, targetWidth - 1.0} };
	
	// the main loop over bounding boxes
	for(int iBb = 0; iBb < numTetragon; ++iBb) {

		double y1 = tetragonData[ iBb ] - 1;
		double x1 = tetragonData[ iBb + numTetragon ] - 1;
		double y2 = tetragonData[ iBb + numTetragon * 2 ] - 1;
		double x2 = tetragonData[ iBb + numTetragon * 3 ] - 1;
		double y3 = tetragonData[ iBb + numTetragon * 4 ] - 1;
		double x3 = tetragonData[ iBb + numTetragon * 5 ] - 1;
		double y4 = tetragonData[ iBb + numTetragon * 6 ] - 1;
		double x4 = tetragonData[ iBb + numTetragon * 7 ] - 1;

		NppiRect sourceRect = {};
    	sourceRect.x = (int) floor(min( min(y1, y2), min(y3, y4) ));
    	sourceRect.y = (int) floor(min( min(x1, x2), min(x3, x4) ));
    	sourceRect.width =  (int) ceil( max( max(y1, y2), max(y3, y4) ) - min( min(y1, y2), min(y3, y4) ) + 1);
    	sourceRect.height = (int) ceil( max( max(x1, x2), max(x3, x4) ) - min( min(x1, x2), min(x3, x4) ) + 1);

    	// adjust bounding box bounds if it is outside of the image
    	if (sourceRect.x < 0) {
    		sourceRect.width = sourceRect.width + sourceRect.x;
    		sourceRect.x = 0.0;
    	}
    	if (sourceRect.y < 0) {
    		sourceRect.height = sourceRect.height + sourceRect.y;
    		sourceRect.y = 0.0;
    	}
    	if (sourceRect.width > imageHeight  - sourceRect.x + 1) {
    		sourceRect.width = imageHeight  - sourceRect.x + 1;
    	}
    	if (sourceRect.height > imageWidth  - sourceRect.y + 1) {
    		sourceRect.height = imageWidth  - sourceRect.y + 1;
    	}

		// double aSrcQuad[4][2] = { {y1 + 0.5, x1 + 0.5}, {y4 + 0.5, x4 + 0.5}, {y3 + 0.5, x3 + 0.5}, {y2 + 0.5, x2 + 0.5} };
		double aSrcQuad[4][2] = { {y1, x1}, {y4, x4}, {y3, x3}, {y2, x2} };

    	float *curOutput = d_outputData + numChannels * targetHeight * targetWidth * iBb;
		const float *pSrc[3] = { d_inputImage, d_inputImage + imageHeight * imageWidth, d_inputImage + 2 * imageHeight * imageWidth};
		float *pDst[3] = { curOutput, curOutput + targetHeight * targetWidth, curOutput + 2 * targetHeight * targetWidth};

        // When NPP_CHECK_NPP catches an error it throws an exception
        // If the exception is not caught, we can get a memory leak on a GPU
        try{
            NppStatus exitCode = nppiWarpPerspectiveQuad_32f_P3R (
                pSrc, // const Npp32f ∗ pSrc[3], 
                nppiImageSize, // NppiSize oSrcSize, 
                imageStep, // int nSrcStep, 
                sourceRect, // NppiRect oSrcROI, 
                aSrcQuad, // const double aSrcQuad[4][2], 
                pDst, // Npp32f ∗ pDst[3], 
                targetStep, // int nDstStep, 
                targetRect, // NppiRect oDstROI, 
                aDstQuad, // const double aDstQuad[4][2], 
                NPPI_INTER_CUBIC //int eInterpolation
                );
            if (exitCode != NPP_SUCCESS) {
                mexPrintf("nppiWarpPerspectiveQuad_32f_P3R returns exit code %d, see http://cseweb.ucsd.edu/classes/wi15/cse262-a/static/cuda-5.5-doc/pdf/NPP_Library.pdf for the description of exit code.\n", exitCode);
                MATLAB_ASSERT(exitCode == NPP_SUCCESS, "cropTetragonsMex: nppiWarpPerspectiveQuad_32f_P3R returns bad exit code");
            }
        } catch (...) {
            // free GPU memory
            mxGPUDestroyGPUArray(outputData);
            mxGPUDestroyGPUArray(inputImage);
            throw;
        }
}
	
	*cropsOutPtr = mxGPUCreateMxArrayOnGPU(outputData);

	// do not forget to free GPU memory
	mxGPUDestroyGPUArray(outputData);
	mxGPUDestroyGPUArray(inputImage);
}
